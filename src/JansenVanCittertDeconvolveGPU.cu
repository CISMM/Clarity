#include "hip/hip_runtime.h"
/* 
 * Clarity is Copyright 2008 Center for Integrated Systems for Microscopy, 
 * Copyright 2008 University of North Carolina at Chapel Hill.
 *
 * Clarity is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Public License as published by the Free Software 
 * Foundation; either version 2 of the License, or (at your option) any 
 * later version.
 *
 * This program is distributed in the hope that it will be useful, but 
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
 * or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License 
 * for more details.
 *
 * You should have received a copy of the GNU General Public License along 
 * with this program; if not, write to the Free Software Foundation, Inc., 
 * 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA. You can also find 
 * the GPL on the GNU web site (http://www.gnu.org/copyleft/gpl.html).
 *
 * File name: JansenVanCittertDeconvolveGPU.cu
 * Author: Cory Quammen <cquammen@cs.unc.edu>
 */


#include <stdio.h>

#define DEFAULT_BLOCKS 64
#define DEFAULT_THREADS_PER_BLOCK 128



#include "JansenVanCittertDeconvolveGPU.h"

extern int getMapBlocks();
extern int getMapThreadsPerBlock();


__global__
void
JansenVanCittertCUDAKernel(
   int n, float* in, float inMax, float invMaxSq, float* i_k,
   float* o_k, float* i_kNext) {

   const int tid     = blockIdx.x*blockDim.x + threadIdx.x;
   const int threadN = blockDim.x*gridDim.x;

   for (int j = tid; j < n; j += threadN) {
      float diff = o_k[j] - inMax;
      float gamma = 1.0f - ((diff * diff) * invMaxSq);
      float val = i_k[j] + (gamma * (in[j] - o_k[j]));
      i_kNext[j] = max(val, 0.0f);
   }
}


extern "C"
void
JansenVanCittertDeconvolveKernelGPU(
   int nx, int ny, int nz, float* in, float inMax, float invMaxSq,
   float* i_k, float* o_k, float* i_kNext) {

   int n = nz*ny*nx;
   dim3 grid(getMapBlocks());
   dim3 block(getMapThreadsPerBlock());

   JansenVanCittertCUDAKernel<<<grid, block>>>(n, in, inMax, invMaxSq, 
      i_k, o_k, i_kNext);

   hipError_t error = hipDeviceSynchronize();
   if (error != hipSuccess) {
      fprintf(stderr, "CUDA error: %s in file '%s' in line %i : %s.\n",
         "JansenVanCittertDeconvolveKernelGPU failed", __FILE__, __LINE__,
              hipGetErrorString(error));
   }
}
