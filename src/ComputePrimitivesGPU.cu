#include "hip/hip_runtime.h"
/* 
 * Clarity is Copyright 2008 Center for Integrated Systems for Microscopy, 
 * Copyright 2008 University of North Carolina at Chapel Hill.
 *
 * Clarity is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Public License as published by the Free Software 
 * Foundation; either version 2 of the License, or (at your option) any 
 * later version.
 *
 * This program is distributed in the hope that it will be useful, but 
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
 * or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License 
 * for more details.
 *
 * You should have received a copy of the GNU General Public License along 
 * with this program; if not, write to the Free Software Foundation, Inc., 
 * 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA. You can also find 
 * the GPL on the GNU web site (http://www.gnu.org/copyleft/gpl.html).
 *
 * File name: ComputePrimitivesGPU.cu
 * Author: Cory Quammen <cquammen@cs.unc.edu>
 */


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "ComputePrimitivesGPU.h"


#define BLOCKS 16
#define THREADS_PER_BLOCK 128


__global__
void
ReduceSumKernelGPU(
   float* blockResults, float* data, int n, int padN) {
   
   extern __shared__ float sdata[];
   int tid  = blockDim.x*blockIdx.x + threadIdx.x;
   int incr = gridDim.x*blockDim.x;
   float sum = 0.0f;

   for (int i = tid; i < padN; i += incr) {
      // Load data into shared memory. All reads 
      // should be coalesced by reading them this way.
      sdata[threadIdx.x] = 0.0f;
      if (i < n)
         sdata[threadIdx.x] = data[i];

      // Reduce the values in shared memory.
      for (int d = blockDim.x >> 1; d > 0; d >>= 1) {
         __syncthreads(); // Make sure all data is read before
                          // proceeding.

         // No bank conflicts in shared memory here.
         if (threadIdx.x < d)
            sdata[threadIdx.x] += sdata[threadIdx.x+d];
      }
      __syncthreads();

      // The reduction results end up in element 0 of shared memory.
      sum += sdata[0];
   }

   // Only thread 0 writes the sum to memory.
   if (threadIdx.x == 0)
      blockResults[blockIdx.x] = sum;
}


extern "C"
void
Clarity_ReduceSumGPU(
   float* result, float* buffer, int n) {
   
   // Set up device call configuration.
   dim3 blockSize(THREADS_PER_BLOCK);
   dim3 gridSize(BLOCKS);
   size_t sharedSize = sizeof(float)*blockSize.x;
   int numThreads = blockSize.x * gridSize.x;
   int paddedArraySize = n;
   int remainder = paddedArraySize % numThreads;
   if (remainder)
      paddedArraySize = ((n / numThreads) + 1) * numThreads;

   // Allocate memory on the device for block-wise partial 
   // reductions computed by the kernel.
   float *blockResultsDev = NULL;
   hipMalloc((void**)&blockResultsDev, sizeof(float)*gridSize.x);

   ReduceSumKernelGPU<<<gridSize, blockSize, sharedSize>>>(
      blockResultsDev, buffer, n, paddedArraySize);

   // Read the partial sums from the blocks back to the host.
   float* blockResultsHost = (float*) malloc(sizeof(float)*gridSize.x);
   hipMemcpy(blockResultsHost, blockResultsDev, 
      sizeof(float)*gridSize.x, hipMemcpyDeviceToHost);

   // Add up the results
   *result = 0.0f;
   for (int i = 0; i < gridSize.x; i++) {
      *result += blockResultsHost[i];
   }

   free(blockResultsHost);
   hipFree(blockResultsDev);
}


__global__
void
MultiplyArraysComponentWiseKernelGPU(
   float* result, float* a, float* b, int n) {

   int tid  = blockDim.x*blockIdx.x + threadIdx.x;
   int incr = gridDim.x*blockDim.x;
   
   for (int i = tid; i < n; i += incr) {
      result[i] = a[i] * b[i];
   }
}


void
Clarity_MultiplyArraysComponentWiseGPU(
   float* result, float* a, float* b, int n) {

   // Set up device call configuration.
   dim3 blockSize(THREADS_PER_BLOCK);
   dim3 gridSize(BLOCKS);

   MultiplyArraysComponentWiseKernelGPU<<<gridSize, blockSize>>>(
      result, a, b, n);
}


__global__
void
DivideArraysComponentWiseKernelGPU(
   float* result, float* a, float* b, float value, int n) {

   int tid  = blockDim.x*blockIdx.x + threadIdx.x;
   int incr = gridDim.x*blockDim.x;
   
   for (int i = tid; i < n; i += incr) {
      if (fabs(b[i]) < 1e-5) {
         result[i] = value;
      } else {
         result[i] = a[i] / b[i];
      }
   }
}


void
Clarity_DivideArraysComponentWiseGPU(
   float* result, float* a, float* b, float value, int n) {

   // Set up device call configuration.
   dim3 blockSize(THREADS_PER_BLOCK);
   dim3 gridSize(BLOCKS);

   DivideArraysComponentWiseKernelGPU<<<gridSize, blockSize>>>(
      result, a, b, value, n);
}


__global__
void
ScaleArrayKernelGPU(
   float* result, float* a, int n, float scale) {

   int tid  = blockDim.x*blockIdx.x + threadIdx.x;
   int incr = gridDim.x*blockDim.x;
   
   for (int i = tid; i < n; i += incr) {
      result[i] = a[i] * scale;
   }
}


extern "C"
void
Clarity_ScaleArrayGPU(
   float* result, float* a, int n, float scale) {

   // Set up device call configuration.
   dim3 blockSize(THREADS_PER_BLOCK);
   dim3 gridSize(BLOCKS);

   ScaleArrayKernelGPU<<<gridSize, blockSize>>>(
      result, a, n, scale);
}
