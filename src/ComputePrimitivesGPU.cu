#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "ComputePrimitivesGPU.h"


#define BLOCKS 16
#define THREADS_PER_BLOCK 128


__global__
void
ReduceSumKernelGPU(
   float* blockResults, float* data, int n, int padN) {
   
   extern __shared__ float sdata[];
   int tid  = blockDim.x*blockIdx.x + threadIdx.x;
   int incr = gridDim.x*blockDim.x;
   float sum = 0.0f;

   for (int i = tid; i < padN; i += incr) {
      // Load data into shared memory. All reads 
      // should be coalesced by reading them this way.
      sdata[threadIdx.x] = 0.0f;
      if (i < n)
         sdata[threadIdx.x] = data[i];

      // Reduce the values in shared memory.
      for (int d = blockDim.x >> 1; d > 0; d >>= 1) {
         __syncthreads(); // Make sure all data is read before
                          // proceeding.

         // No bank conflicts in shared memory here.
         if (threadIdx.x < d)
            sdata[threadIdx.x] += sdata[threadIdx.x+d];
      }
      __syncthreads();

      // The reduction results end up in element 0 of shared memory.
      sum += sdata[0];
   }

   // Only thread 0 writes the sum to memory.
   if (threadIdx.x == 0)
      blockResults[blockIdx.x] = sum;
}


extern "C"
void
Clarity_ReduceSumGPU(
   float* result, float* buffer, int n) {
   
   // Set up device call configuration.
   dim3 blockSize(THREADS_PER_BLOCK);
   dim3 gridSize(BLOCKS);
   size_t sharedSize = sizeof(float)*blockSize.x;
   int numThreads = blockSize.x * gridSize.x;
   int paddedArraySize = n;
   int remainder = paddedArraySize % numThreads;
   if (remainder)
      paddedArraySize = ((n / numThreads) + 1) * numThreads;

   // Allocate memory on the device for block-wise partial 
   // reductions computed by the kernel.
   float *blockResultsDev = NULL;
   hipMalloc((void**)&blockResultsDev, sizeof(float)*gridSize.x);

   ReduceSumKernelGPU<<<gridSize, blockSize, sharedSize>>>(
      blockResultsDev, buffer, n, paddedArraySize);

   // Read the partial sums from the blocks back to the host.
   float* blockResultsHost = (float*) malloc(sizeof(float)*gridSize.x);
   hipMemcpy(blockResultsHost, blockResultsDev, 
      sizeof(float)*gridSize.x, hipMemcpyDeviceToHost);

   // Add up the results
   *result = 0.0f;
   for (int i = 0; i < gridSize.x; i++) {
      *result += blockResultsHost[i];
   }

   free(blockResultsHost);
   hipFree(blockResultsDev);
}


__global__
void
MultiplyArraysComponentWiseKernelGPU(
   float* result, float* a, float* b, int n) {

   int tid  = blockDim.x*blockIdx.x + threadIdx.x;
   int incr = gridDim.x*blockDim.x;
   
   for (int i = tid; i < n; i += incr) {
      result[i] = a[i] * b[i];
   }
}


void
Clarity_MultiplyArraysComponentWiseGPU(
   float* result, float* a, float* b, int n) {

   // Set up device call configuration.
   dim3 blockSize(THREADS_PER_BLOCK);
   dim3 gridSize(BLOCKS);

   MultiplyArraysComponentWiseKernelGPU<<<gridSize, blockSize>>>(
      result, a, b, n);
}


__global__
void
DivideArraysComponentWiseKernelGPU(
   float* result, float* a, float* b, float value, int n) {

   int tid  = blockDim.x*blockIdx.x + threadIdx.x;
   int incr = gridDim.x*blockDim.x;
   
   for (int i = tid; i < n; i += incr) {
      if (fabs(b[i]) < 1e-5) {
         result[i] = value;
      } else {
         result[i] = a[i] / b[i];
      }
   }
}


void
Clarity_DivideArraysComponentWiseGPU(
   float* result, float* a, float* b, float value, int n) {

   // Set up device call configuration.
   dim3 blockSize(THREADS_PER_BLOCK);
   dim3 gridSize(BLOCKS);

   DivideArraysComponentWiseKernelGPU<<<gridSize, blockSize>>>(
      result, a, b, value, n);
}


__global__
void
ScaleArrayKernelGPU(
   float* result, float* a, int n, float scale) {

   int tid  = blockDim.x*blockIdx.x + threadIdx.x;
   int incr = gridDim.x*blockDim.x;
   
   for (int i = tid; i < n; i += incr) {
      result[i] = a[i] * scale;
   }
}


extern "C"
void
Clarity_ScaleArrayGPU(
   float* result, float* a, int n, float scale) {

   // Set up device call configuration.
   dim3 blockSize(THREADS_PER_BLOCK);
   dim3 gridSize(BLOCKS);

   ScaleArrayKernelGPU<<<gridSize, blockSize>>>(
      result, a, n, scale);
}