#include "hip/hip_runtime.h"
/* 
 * Clarity is Copyright 2008 Center for Integrated Systems for Microscopy, 
 * Copyright 2008 University of North Carolina at Chapel Hill.
 *
 * Clarity is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Public License as published by the Free Software 
 * Foundation; either version 2 of the License, or (at your option) any 
 * later version.
 *
 * This program is distributed in the hope that it will be useful, but 
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
 * or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License 
 * for more details.
 *
 * You should have received a copy of the GNU General Public License along 
 * with this program; if not, write to the Free Software Foundation, Inc., 
 * 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA. You can also find 
 * the GPL on the GNU web site (http://www.gnu.org/copyleft/gpl.html).
 *
 * File name: ComputePrimitivesGPU.cu
 * Author: Cory Quammen <cquammen@cs.unc.edu>
 */


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "ComputePrimitivesGPU.h"


#define DEFAULT_BLOCKS 64
#define DEFAULT_THREADS_PER_BLOCK 128

#define CLARITY_REDUCE_BLOCKS_ENV            "CLARITY_REDUCE_BLOCKS"
#define CLARITY_REDUCE_THREADS_PER_BLOCK_ENV "CLARITY_REDUCE_THREADS_PER_BLOCK"

int getReduceBlocks() {
  int numBlocks = DEFAULT_BLOCKS;
  char *blocksString = getenv(CLARITY_REDUCE_BLOCKS_ENV);
  if (blocksString) {
    numBlocks = atoi(blocksString);
  }
  
  return numBlocks;
}


int getReduceThreadsPerBlock() {
  int numThreadsPerBlock = DEFAULT_THREADS_PER_BLOCK;
  char *threadsPerBlockString = getenv(CLARITY_REDUCE_THREADS_PER_BLOCK_ENV);
  if (threadsPerBlockString) {
    numThreadsPerBlock = atoi(threadsPerBlockString);
  }
  
  return numThreadsPerBlock;
}


#define CLARITY_MAP_BLOCKS_ENV            "CLARITY_MAP_BLOCKS"
#define CLARITY_MAP_THREADS_PER_BLOCK_ENV "CLARITY_MAP_THREADS_PER_BLOCK"

int getMapBlocks() {
  int numBlocks = DEFAULT_BLOCKS;
  char *blockString = getenv(CLARITY_MAP_BLOCKS_ENV);
  if (blockString) {
    numBlocks = atoi(blockString);
  }

  return numBlocks;
}


int getMapThreadsPerBlock() {
  int numThreadsPerBlock = DEFAULT_THREADS_PER_BLOCK;
  char *threadsPerBlockString = getenv(CLARITY_MAP_THREADS_PER_BLOCK_ENV);
  if (threadsPerBlockString) {
    numThreadsPerBlock = atoi(threadsPerBlockString);
  }

  return numThreadsPerBlock;
}


__global__
void
ReduceSumKernelGPU(float* blockResults, float* data, int n) {
  
  extern __shared__ float accumulator[];
  int tid  = blockDim.x*blockIdx.x + threadIdx.x;
  int incr = gridDim.x*blockDim.x;

  accumulator[threadIdx.x] = 0.0f;
    
  for (int i = tid; i < n; i += incr) {
    // All reads should be coalesced with this pattern.
    accumulator[threadIdx.x] += data[i];
  }
    
  // Reduce the values in shared memory.
  for (int d = blockDim.x >> 1; d > 0; d >>= 1) {
    __syncthreads(); // Make sure all data is read before moving on.
      
    // No bank conflicts in shared memory here.
    if (threadIdx.x < d)
      accumulator[threadIdx.x] += accumulator[threadIdx.x+d];
  }
  __syncthreads();
    
  // Only thread 0 writes the sum to memory.
  if (threadIdx.x == 0)
    blockResults[blockIdx.x] = accumulator[0];
}


extern "C"
void
Clarity_ReduceSumGPU(float* result, float* buffer, int n) {
  
  // Set up device call configuration.
  dim3 gridSize(getReduceBlocks());
  dim3 blockSize(getReduceThreadsPerBlock());
  size_t sharedSize = sizeof(float)*blockSize.x;
  
  // Allocate memory on the device for block-wise partial 
  // reductions computed by the kernel.
  float *blockResultsDev = NULL;
  hipMalloc((void**)&blockResultsDev, sizeof(float)*gridSize.x);
  
  ReduceSumKernelGPU<<<gridSize, blockSize, sharedSize>>>
    (blockResultsDev, buffer, n);
  
  // Read the partial sums from the blocks back to the host.
  float* blockResultsHost = (float*) malloc(sizeof(float)*gridSize.x);
  hipMemcpy(blockResultsHost, blockResultsDev, 
	     sizeof(float)*gridSize.x, hipMemcpyDeviceToHost);
  
  // Add up the results
  *result = 0.0f;
  for (int i = 0; i < gridSize.x; i++) {
    *result += blockResultsHost[i];
  }
  
  free(blockResultsHost);
  hipFree(blockResultsDev);
}


__global__
void
MultiplyArraysComponentWiseKernelGPU(float* result, float* a, float* b, int n) {
  int tid  = blockDim.x*blockIdx.x + threadIdx.x;
  int incr = gridDim.x*blockDim.x;
  
  for (int i = tid; i < n; i += incr) {
    result[i] = a[i] * b[i];
  }
}


void
Clarity_MultiplyArraysComponentWiseGPU(float* result, float* a, float* b, int n) {
  
  // Set up device call configuration.
  dim3 gridSize(getMapBlocks());
  dim3 blockSize(getMapThreadsPerBlock());

  MultiplyArraysComponentWiseKernelGPU<<<gridSize, blockSize>>>
    (result, a, b, n);
}


__global__
void
DivideArraysComponentWiseKernelGPU(float* result, float* a, float* b, float value, int n) {
  
  int tid  = blockDim.x*blockIdx.x + threadIdx.x;
  int incr = gridDim.x*blockDim.x;
  
  for (int i = tid; i < n; i += incr) {
    if (fabs(b[i]) < 1e-5) {
      result[i] = value;
    } else {
      result[i] = a[i] / b[i];
    }
  }
}


void
Clarity_DivideArraysComponentWiseGPU(float* result, float* a, float* b, float value, int n) {
  
  // Set up device call configuration.
  dim3 gridSize(getMapBlocks());
  dim3 blockSize(getMapThreadsPerBlock());
  
  DivideArraysComponentWiseKernelGPU<<<gridSize, blockSize>>>
    (result, a, b, value, n);

  hipError_t error = hipDeviceSynchronize();
  if (error != hipSuccess) {
    fprintf(stderr, "CUDA error: %s in file '%s' in line %i : %s.\n",
            "Clarity_DivideArraysComponentWiseGPU failed", __FILE__, __LINE__,
            hipGetErrorString(error));
  }

}


__global__
void
ScaleArrayKernelGPU(float* result, float* a, int n, float scale) {

  int tid  = blockDim.x*blockIdx.x + threadIdx.x;
  int incr = gridDim.x*blockDim.x;
  
  for (int i = tid; i < n; i += incr) {
    result[i] = a[i] * scale;
  }
}


extern "C"
void
Clarity_ScaleArrayGPU(float* result, float* a, int n, float scale) {
  
  // Set up device call configuration.
  dim3 gridSize(getMapBlocks());
  dim3 blockSize(getMapThreadsPerBlock());
  
  ScaleArrayKernelGPU<<<gridSize, blockSize>>>
    (result, a, n, scale);

  hipError_t error = hipDeviceSynchronize();
  if (error != hipSuccess) {
    fprintf(stderr, "CUDA error: %s in file '%s' in line %i : %s.\n",
            "Clarity_ScaleArrayGPU failed", __FILE__, __LINE__,
            hipGetErrorString(error));
  }
}
