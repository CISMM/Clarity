#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "ComplexCUDA.h"

#define BLOCKS 16
#define THREADS_PER_BLOCK 128


__global__ void WienerCUDAKernel(int n, float scale, Complex* inFT,
                                 Complex* psfFT, Complex* outFT,
                                 float sigma, float epsilon) {
   const int tid     = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
   const int threadN = __mul24(blockDim.x, gridDim.x);

   for (int voxelID = tid; voxelID < n; voxelID += threadN) {
      Complex H = psfFT[voxelID];
      Complex HConj = complexConjugate(H);
      float HMagSquared = complexMagnitudeSquared(H);
      HConj = complexScale(HConj, 1.0f / (HMagSquared + epsilon));
      outFT[voxelID] = complexMulAndScale(HConj, inFT[voxelID], scale);
   }
}


extern "C"
void
WienerDeconvolveKernelGPU(int nx, int ny, int nz, float* inFT, float* psfFT, 
                    float* outFT, float sigma, float epsilon) {
   int n = nz*ny*(nx/2 + 1);
   dim3 grid(BLOCKS);
   dim3 block(THREADS_PER_BLOCK);
   float scale = 1.0f / ((float) nx*ny*nz);

   WienerCUDAKernel<<<grid, block>>>(n, scale, (Complex*)inFT,
      (Complex*)psfFT, (Complex*)outFT, sigma, epsilon);

   hipError_t result = hipDeviceSynchronize();
   if (result != hipSuccess) {
      fprintf(stderr, "CUDA error: %s in file '%s' in line %i : %s.\n",
         "WienerCUDAKernel failed", __FILE__, __LINE__, hipGetErrorString(result));
   }
}
