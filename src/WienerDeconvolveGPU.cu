#include "hip/hip_runtime.h"
/* 
 * Clarity is Copyright 2008 Center for Integrated Systems for Microscopy, 
 * Copyright 2008 University of North Carolina at Chapel Hill.
 *
 * Clarity is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Public License as published by the Free Software 
 * Foundation; either version 2 of the License, or (at your option) any 
 * later version.
 *
 * This program is distributed in the hope that it will be useful, but 
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
 * or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License 
 * for more details.
 *
 * You should have received a copy of the GNU General Public License along 
 * with this program; if not, write to the Free Software Foundation, Inc., 
 * 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA. You can also find 
 * the GPL on the GNU web site (http://www.gnu.org/copyleft/gpl.html).
 *
 * File name: WienerDeconvolveGPU.cu
 * Author: Cory Quammen <cquammen@cs.unc.edu>
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#include "ComplexCUDA.h"
#include "WienerDeconvolveGPU.h"

extern int getMapBlocks();
extern int getMapThreadsPerBlock();

__global__
void
WienerCUDAKernel(
   int n, float scale, Complex* inFT, Complex* psfFT, 
   Complex* outFT, float epsilon) {

   const int tid     = blockIdx.x*blockDim.x + threadIdx.x;
   const int threadN = blockDim.x*gridDim.x;

   for (int voxelID = tid; voxelID < n; voxelID += threadN) {
      Complex H = psfFT[voxelID];
      Complex HConj = ComplexConjugate(H);
      float HMagSquared = ComplexMagnitudeSquared(H);
      HConj = ComplexScale(HConj, 1.0f / (HMagSquared + epsilon));
      outFT[voxelID] = ComplexMultiplyAndScale(
         HConj, inFT[voxelID], scale);
   }
}


extern "C"
void
WienerDeconvolveKernelGPU(
   int nx, int ny, int nz, float* inFT, float* psfFT, 
   float* outFT, float epsilon) {

   int n = nz*ny*(nx/2 + 1);
   dim3 grid(getMapBlocks());
   dim3 block(getMapThreadsPerBlock());
   float scale = 1.0f / ((float) nx*ny*nz);

   WienerCUDAKernel<<<grid, block>>>(n, scale, (Complex*)inFT,
      (Complex*)psfFT, (Complex*)outFT, epsilon);

   hipError_t result = hipDeviceSynchronize();
   if (result != hipSuccess) {
      fprintf(stderr, "CUDA error: %s in file '%s' in line %i : %s.\n",
         "WienerCUDAKernel failed", __FILE__, __LINE__, hipGetErrorString(result));
   }
}
