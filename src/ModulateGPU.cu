#include "hip/hip_runtime.h"
/* 
 * Clarity is Copyright 2008 Center for Integrated Systems for Microscopy, 
 * Copyright 2008 University of North Carolina at Chapel Hill.
 *
 * Clarity is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Public License as published by the Free Software 
 * Foundation; either version 2 of the License, or (at your option) any 
 * later version.
 *
 * This program is distributed in the hope that it will be useful, but 
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
 * or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License 
 * for more details.
 *
 * You should have received a copy of the GNU General Public License along 
 * with this program; if not, write to the Free Software Foundation, Inc., 
 * 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA. You can also find 
 * the GPL on the GNU web site (http://www.gnu.org/copyleft/gpl.html).
 *
 * File name: FFTGPU.cu
 * Author: Cory Quammen <cquammen@cs.unc.edu>
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#include "ComplexCUDA.h"

#define DEFAULT_BLOCKS 64
#define DEFAULT_THREADS_PER_BLOCK 128

#define CLARITY_MODULATE_BLOCKS_ENV            "CLARITY_MODULATE__BLOCKS"
#define CLARITY_MODULATE_THREADS_PER_BLOCK_ENV "CLARITY_MODULATE_THREADS_PER_BLOCK"

int getModulateBlocks() {
  int numBlocks = DEFAULT_BLOCKS;
  char *blockString = getenv(CLARITY_MODULATE_BLOCKS_ENV);
  if (blockString) {
    numBlocks = atoi(blockString);
  }

  return numBlocks;
}


int getModulateThreadsPerBlock() {
  int numThreadsPerBlock = DEFAULT_THREADS_PER_BLOCK;
  char *threadsPerBlockString = getenv(CLARITY_MODULATE_THREADS_PER_BLOCK_ENV);
  if (threadsPerBlockString) {
    numThreadsPerBlock = atoi(threadsPerBlockString);
  }

  return numThreadsPerBlock;
}



__global__ void ModulateCUDAKernel(int n, float scale, Complex* inFT, Complex* psfFT, Complex* outFT) {
   const int tid     = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
   const int threadN = __mul24(blockDim.x, gridDim.x);

   for (int voxelID = tid; voxelID < n; voxelID += threadN) {
      outFT[voxelID] = ComplexMultiplyAndScale(inFT[voxelID], psfFT[voxelID],
					       scale);
   }

}


extern "C"
void
Clarity_Modulate_KernelGPU(int nx, int ny, int nz, float* inFT,
                           float* psfFT, float* outFT) {
   int n = nz*ny*(nx/2 + 1);
   dim3 grid(getModulateBlocks());
   dim3 block(getModulateThreadsPerBlock());
   float scale = 1.0f / ((float) nx*ny*nz);

   ModulateCUDAKernel<<<grid, block>>>(n, scale, (Complex *) inFT, 
				       (Complex *) psfFT, (Complex *) outFT);

   hipError_t result = hipDeviceSynchronize();
   if (result != hipSuccess) {
      fprintf(stderr, "CUDA error: %s in file '%s' in line %i : %s.\n",
              "ModulateCUDAKernel failed", __FILE__, __LINE__, hipGetErrorString(result));

   }
}
